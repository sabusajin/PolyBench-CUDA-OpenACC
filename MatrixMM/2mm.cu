
#include <iostream>
#include <chrono>
#include "hip/hip_runtime.h"


static void HandleError( hipError_t err, const char *file, int line ) 
{
    if (err != hipSuccess) 
    {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#define SIZE 10

using namespace std;

void print_array (float array[][SIZE]) 
{

	for (int i=0; i<SIZE; i++) {
		for (int j=0; j<SIZE; j++) {
			cout<<array[i][j]<<" ";
		}
		cout<<endl;
	}
}

void initialize_array (float array[][SIZE]) 
{

	for (int i=0; i<SIZE; i++) {
		for (int j=0; j<SIZE; j++) {
			array[i][j] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
		}
	}
}

void array_decl () 
{

	static float A[SIZE][SIZE];
	static float B[SIZE][SIZE];
	static float C[SIZE][SIZE];
	static float D[SIZE][SIZE];

	initialize_array(A);
	initialize_array(B);
	initialize_array(C);
	initialize_array(D);

	int *d_A, *d_B, *d_C, *d_D, *d_temp;

	HANDLE_ERROR(hipMalloc((void**)&d_A, SIZE*SIZE*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_B, SIZE*SIZE*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_C, SIZE*SIZE*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_D, SIZE*SIZE*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_temp, SIZE*SIZE*sizeof(float)));

	hipFree (d_A);
	hipFree (d_B);
	hipFree (d_C);
	hipFree (d_D);
	hipFree (d_temp);

}

int main (int argc, char **argv) {

	
	array_decl();







}